#include<hip/hip_runtime.h>

__global__ void kernel(int* a, int* b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

void test_kernel(int* a_, int* b_)
{
	int* a;
	int* b;
	hipMallocManaged(&a, sizeof(int)*10);
	hipMallocManaged(&b, sizeof(int)*10);
	for(int i = 0; i < 10; i++) {
		a[i] = a_[i];
		b[i] = b_[i];
	}
	kernel<<< 1, 10>>>(a, b);
	hipDeviceSynchronize();
	for(int i = 0; i < 10; i++) {
                a_[i] = a[i];
                b_[i] = b[i];
        }
}
